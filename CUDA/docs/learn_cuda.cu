#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>



/**
 * HOST: handle the CUDA Errors.
 */
#define HANDLE_CUDA_ERROR( cuda_expression ) { assertGpuError( ( cuda_expression ), __FILE__, __LINE__ ); }

inline void assertGpuError( hipError_t error_index, const char *error_file, const unsigned error_line ) {
	if ( error_index != hipSuccess ) {
		fprintf( stderr, "\n\n\n***\nCUDA ERROR :: %s [LINE %u] ---> %s.\n***\n\n\n",
				error_file, error_line, hipGetErrorString( error_index ) );
		hipDeviceReset();
		exit( EXIT_FAILURE );
	}
}



/**
 * DEVICE: add two vectors.
 */
__global__ void devAddVect( const double *dev_vect_a, const double *dev_vect_b,
		double *dev_vect_c, const unsigned vect_length ) {
	unsigned tidx = threadIdx.x + blockIdx.x * blockDim.x;
	if ( tidx >= vect_length ) {
		return ;
	}
	dev_vect_c[ tidx ] = dev_vect_a[ tidx ] + dev_vect_b[ tidx ];
	__syncthreads();
}



/**
 * DEVICE: add two vectors.
 */
__global__ void devAddVectArbitraryLength( const double *dev_vect_a, const double *dev_vect_b,
		double *dev_vect_c, const unsigned vect_length ) {
	unsigned tidx = threadIdx.x + blockIdx.x * blockDim.x;
	while ( tidx < vect_length ) {
		dev_vect_c[ tidx ] = dev_vect_a[ tidx ] + dev_vect_b[ tidx ];
		tidx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}



/**
 * HOST: whether two double-precision floating-point values are approximately equal or not.
 */
int is_equal( double x, double y ) {
	return fabs( x - y ) < 1e-6 ? 1 : 0;
}



void call_devAddVect( void ) {
	/* add two vectors */
	printf( "\n*********************\n* add two vectors *\n*********************\n" );
	int ind_dev = 0;
	HANDLE_CUDA_ERROR( hipSetDevice( ind_dev ) );

	unsigned vect_length = 10000000;
	size_t vect_length_bytes = vect_length * sizeof( double );
	double *vect_a = NULL, *vect_b = NULL, *vect_c = NULL;
	vect_a = ( double * ) malloc( vect_length_bytes );
	vect_b = ( double * ) malloc( vect_length_bytes );
	vect_c = ( double * ) malloc( vect_length_bytes );
	if ( vect_a == NULL || vect_b == NULL || vect_c == NULL ) {
		fprintf( stderr, "\nHOST ERROR :: cannot allocate enough memory.\n" );
		exit( EXIT_FAILURE );
	}
	for ( unsigned i = 0; i < vect_length; i++ ) {
		vect_a[ i ] = i;
		vect_b[ i ] = 2 * i;
	}

	double *dev_vect_a, *dev_vect_b, *dev_vect_c;
	HANDLE_CUDA_ERROR( hipMalloc( ( double ** ) &dev_vect_a, vect_length_bytes ) );
	HANDLE_CUDA_ERROR( hipMalloc( ( double ** ) &dev_vect_b, vect_length_bytes ) );
	HANDLE_CUDA_ERROR( hipMalloc( ( double ** ) &dev_vect_c, vect_length_bytes ) );

	HANDLE_CUDA_ERROR( hipMemcpy( dev_vect_a, vect_a, vect_length_bytes, hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( dev_vect_b, vect_b, vect_length_bytes, hipMemcpyHostToDevice ) );

	unsigned num_thread_per_block = 512;
	unsigned num_block_per_grid = (vect_length + num_thread_per_block - 1 ) / num_thread_per_block;
	time_t gpu_run_time_start, gpu_run_time_end;
	time( &gpu_run_time_start );
	devAddVect <<< num_block_per_grid, num_thread_per_block >>> (
			dev_vect_a, dev_vect_b, dev_vect_c, vect_length );
	time( &gpu_run_time_end );
	double gpu_run_time = difftime( gpu_run_time_end, gpu_run_time_start );
	HANDLE_CUDA_ERROR( hipPeekAtLastError() );
	HANDLE_CUDA_ERROR( hipMemcpy( vect_c, dev_vect_c, vect_length_bytes, hipMemcpyDeviceToHost ) );

	printf( "Totoal GPU run time: %lf.\n", gpu_run_time );
	for ( unsigned i = 0; i < vect_length; i++ ) {
		if ( is_equal( vect_c[ i ], vect_a[ i ] + vect_b[ i ] ) == 0 ) {
			printf( "\nERROR :: final results are not right.\n" );
			printf( ">> i = %u :: vect_c[ i ] = %lf vs. vect_a[ i ] + vect_b[ i ] = %lf.\n",
					i, vect_c[ i ], vect_a[ i ] + vect_b[ i ] );
			exit( EXIT_FAILURE );
		} else {
			if ( i == 0 || i == ( vect_length / 2) || i == ( vect_length - 1 ) ) {
				printf( ">> i = %u :: vect_c (%.2lf) == vect_a (%.2lf) + vect_b (%.2lf).\n",
						i, vect_c[ i ], vect_a[ i ], vect_b[ i ] );
			}
		}
	}

	HANDLE_CUDA_ERROR( hipFree( dev_vect_a ) );
	HANDLE_CUDA_ERROR( hipFree( dev_vect_b ) );
	HANDLE_CUDA_ERROR( hipFree( dev_vect_c ) );
	free( vect_a );
	free( vect_b );
	free( vect_c );
	hipDeviceReset();
}



void call_devAddVectArbitraryLength( void ) {
	/* add two vectors with arbitrary length */
		printf( "\n*********************\n* add two vectors with arbitrary length *\n*********************\n" );
		int ind_dev = 1;
		HANDLE_CUDA_ERROR( hipSetDevice( ind_dev) );

		unsigned vect_length = 100000000;
		size_t vect_length_bytes = vect_length * sizeof( double );
		double *vect_a = NULL, *vect_b = NULL, *vect_c = NULL;
		vect_a = ( double * ) malloc( vect_length_bytes );
		vect_b = ( double * ) malloc( vect_length_bytes );
		vect_c = ( double * ) malloc( vect_length_bytes );
		if ( vect_a == NULL || vect_b == NULL || vect_c == NULL ) {
			fprintf( stderr, "\nHOST ERROR :: cannot allocate enough memory.\n" );
		}
		for ( unsigned i = 0; i < vect_length; i++ ) {
			vect_a[ i ] = i;
			vect_b[ i ] = 2 * i;
		}

		double *dev_vect_a, *dev_vect_b, *dev_vect_c;
		HANDLE_CUDA_ERROR( hipMalloc( ( double ** ) &dev_vect_a, vect_length_bytes ) );
		HANDLE_CUDA_ERROR( hipMalloc( ( double ** ) &dev_vect_b, vect_length_bytes ) );
		HANDLE_CUDA_ERROR( hipMalloc( ( double ** ) &dev_vect_c, vect_length_bytes ) );

		HANDLE_CUDA_ERROR( hipMemcpy( dev_vect_a, vect_a, vect_length_bytes, hipMemcpyHostToDevice ) );
		HANDLE_CUDA_ERROR( hipMemcpy( dev_vect_b, vect_b, vect_length_bytes, hipMemcpyHostToDevice ) );

		unsigned num_thread_per_block = 1024;
		unsigned num_block_per_grid = 1024;
		time_t gpu_run_time_start, gpu_run_time_end;
		time( &gpu_run_time_start );
		devAddVectArbitraryLength <<< num_block_per_grid, num_thread_per_block >>> (
				dev_vect_a, dev_vect_b, dev_vect_c, vect_length );
		time( &gpu_run_time_end );
		double gpu_run_time = difftime( gpu_run_time_end, gpu_run_time_start );
		HANDLE_CUDA_ERROR( hipPeekAtLastError() );
		HANDLE_CUDA_ERROR( hipMemcpy( vect_c, dev_vect_c, vect_length_bytes, hipMemcpyDeviceToHost ) );

		printf( "Totoal GPU run time: %lf.\n", gpu_run_time );
		for ( unsigned i = 0; i < vect_length; i++ ) {
			if ( is_equal( vect_c[ i ], vect_a[ i ] + vect_b[ i ] ) == 0 ) {
				printf( "\nError :: final results are not right.\n" );
				printf( ">> i = %u :: vect_c[ i ] = %lf vs. vect_a[ i ] + vect_b[ i ] = %lf.\n",
						i, vect_c[ i ], vect_a[ i ] + vect_b[ i ] );
				exit( EXIT_FAILURE );
			} else {
				if ( i == 0 || i == ( vect_length / 2 ) || i == ( vect_length - 1 ) ) {
					printf( ">> i = %u :: vect_c (%.2lf) == vect_a (%.2lf) + vect_b (%.2lf).\n",
							i, vect_c[ i ], vect_a[ i ], vect_b[ i ] );
				}
			}
		}

		HANDLE_CUDA_ERROR( hipFree( dev_vect_a ) );
		HANDLE_CUDA_ERROR( hipFree( dev_vect_b ) );
		HANDLE_CUDA_ERROR( hipFree( dev_vect_c ) );
		free( vect_a );
		free( vect_b );
		free( vect_c );
		hipDeviceReset();
}



int main( void ) {
	call_devAddVect();
	call_devAddVectArbitraryLength();
}
